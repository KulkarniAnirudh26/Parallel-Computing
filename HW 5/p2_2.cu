
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <hipblas.h>
#include <time.h>

#define size 1024
#define blocksize 32

__global__ void mat_mul(int *a, int *b, int *c){
	int my_x;
	int my_y;
	int row = threadIdx.y;
	int col = threadIdx.x;
	my_x = blockIdx.x*blockDim.x + threadIdx.x;
	my_y = blockIdx.y*blockDim.y + threadIdx.y;

	__shared__ float A_share[blocksize][blocksize];
	__shared__ float B_share[blocksize][blocksize];

	int local_c = 0;

	for(int i = 0; i < size/blocksize;i++){
		A_share[row][col] = a[my_x*size + (i*blockDim.y + col)];
		B_share[row][col] = b[(i*blockDim.x + row)*size + my_y];
		__syncthreads();
		for(int j =0; j< blocksize;j++){
			local_c += A_share[row][j]*B_share[j][col]; 
		}
		__syncthreads();
	}
	c[my_x*size + my_y] = local_c;  
}

int main(){		
    int i;
    int *a = (int*)malloc(sizeof(int)*size*size);          
    int *b = (int*)malloc(sizeof(int)*size*size);          
    int *c = (int*)malloc(sizeof(int)*size*size);           	
	
	  for(i=0; i<size*size; i++){
			a[i]=1;
			b[i]=2;
  	}
		int *gpu_a, *gpu_b, *gpu_c;
		hipMalloc((void**)&gpu_a, sizeof(int)*size*size); 
		hipMalloc((void**)&gpu_b, sizeof(int)*size*size);
		hipMalloc((void**)&gpu_c, sizeof(int)*size*size);
		
		struct timespec start, stop; 
	    double time;
	  
	  
		hipMemcpy(gpu_a, a, sizeof(int)*size*size, hipMemcpyHostToDevice);
		hipMemcpy(gpu_b, b, sizeof(int)*size*size, hipMemcpyHostToDevice);
		
		dim3 dimGrid(32,32);
		dim3 dimBlock(32,32);
		
		if( clock_gettime( CLOCK_REALTIME, &start) == -1 ) { perror( "clock gettime" );}
		mat_mul<<<dimGrid, dimBlock>>>(gpu_a, gpu_b, gpu_c);				
		hipMemcpy(c, gpu_c, sizeof(int)*size*size, hipMemcpyDeviceToHost);
		
		if( clock_gettime( CLOCK_REALTIME, &stop) == -1 ) { perror( "clock gettime" );}	  
		time = (stop.tv_sec - start.tv_sec)+ (double)(stop.tv_nsec - start.tv_nsec)/1e9;
		printf("time is %f ns\n", time*1e9);	 
		
		printf("C[%d][%d] = %d ",451,451,c[451*size +451]);
  	
		free(a);
		free(b);
		free(c);
		hipFree(gpu_a);  
		hipFree(gpu_b);  
		hipFree(gpu_c);  
		return 0;
}	
