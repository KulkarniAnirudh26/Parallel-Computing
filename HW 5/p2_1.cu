
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <hipblas.h>
#include <time.h>

#define size 1024


__global__ void mat_mul(int *a, int *b, int *c){
	int my_x,my_y;
	my_x = blockIdx.x*blockDim.x + threadIdx.x;
	my_y = blockIdx.y*blockDim.y + threadIdx.y;
	int local_c = 0;
	for(int i =0; i< size;i++){
		local_c = local_c +  a[my_x*size + i]*b[i*size + my_y]; 
	}
	c[my_x*size + my_y] = local_c;  
}

int main(){		
    int i;
    int *a = (int*)malloc(sizeof(int)*size*size);          
	int *b = (int*)malloc(sizeof(int)*size*size);          
    int *c = (int*)malloc(sizeof(int)*size*size);           	
	
	  for(i=0; i<size*size; i++){
			a[i]=1;
			b[i]=2;
  	}
		int *gpu_a, *gpu_b, *gpu_c;
		hipMalloc((void**)&gpu_a, sizeof(int)*size*size); 
		hipMalloc((void**)&gpu_b, sizeof(int)*size*size);
		hipMalloc((void**)&gpu_c, sizeof(int)*size*size);
		
		struct timespec start, stop; 
	    double time;
	  
	  
		hipMemcpy(gpu_a, a, sizeof(int)*size*size, hipMemcpyHostToDevice);
		hipMemcpy(gpu_b, b, sizeof(int)*size*size, hipMemcpyHostToDevice);
		
		dim3 dimGrid(64,64);
		dim3 dimBlock(16,16);
		
		if( clock_gettime( CLOCK_REALTIME, &start) == -1 ) { perror( "clock gettime" );}
		mat_mul<<<dimGrid, dimBlock>>>(gpu_a, gpu_b, gpu_c);				
		hipMemcpy(c, gpu_c, sizeof(int)*size*size, hipMemcpyDeviceToHost);
		
		if( clock_gettime( CLOCK_REALTIME, &stop) == -1 ) { perror( "clock gettime" );}	  
		time = (stop.tv_sec - start.tv_sec)+ (double)(stop.tv_nsec - start.tv_nsec)/1e9;
		printf("time is %f ns\n", time*1e9);	 
		
		printf("C[%d][%d] = %d ",451,451,c[451*size +451]);
  	
		free(a);
		free(b);
		free(c);
		hipFree(gpu_a);  
		hipFree(gpu_b);  
		hipFree(gpu_c);  
		return 0;
}	
